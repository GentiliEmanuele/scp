#include "cuda_time.h"
#include "cuda_mtx.h"
#include "hll.h"
#include "spmv_cuda.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define pr_err(err) printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err))

int hll_time(const char *path, int runs_num, int hack_size, struct time_info *ti) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct hll sm;
    if (hll_init(&sm, hack_size, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *v = d_random(mm.num_rows);
    double *d_data;
    int *d_col_index;
    int *d_maxnzr;
    int *d_offsets;
    hipError_t err = cuda_hll_init(&sm, &d_data, &d_col_index, &d_maxnzr, &d_offsets);
    if (err != hipSuccess) {
        pr_err(err);
        hll_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < runs_num; ++i) {
        hipEventRecord(start);
        cuda_spmv_hll<<<2, 1024>>>(d_result, sm.hack_size, sm.hacks_num, d_data, d_offsets, d_col_index, d_maxnzr, d_v, sm.num_rows);
        hipEventRecord(stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            pr_err(err);
            hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_maxnzr);
            hipFree(d_result);
            hipFree(d_v);
            hll_cleanup(&sm);
            return -1;
        }
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        sum += m;
    }
    ti->millis = sum / runs_num;
    ti->flops = (2 * nz) / ti->millis;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_maxnzr);
    hipFree(d_result);
    hipFree(d_v);
    free(v);
    hll_cleanup(&sm);
    return 0;
}