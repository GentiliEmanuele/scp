#include "hip/hip_runtime.h"
#include "cuda_time.h"
#include "cuda_mtx.h"
#include "hll.h"
#include "spmv_cuda.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define pr_err(err) printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err))

int hll_time(const char *path, int runs_num, int hack_size, struct time_info *ti) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct hll sm;
    if (hll_init(&sm, hack_size, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *v = d_random(mm.num_rows);
    double *d_data;
    int *d_col_index;
    int *d_maxnzr;
    int *d_offsets;
    hipError_t err = cuda_hll_init(&sm, &d_data, &d_col_index, &d_maxnzr, &d_offsets);
    if (err != hipSuccess) {
        pr_err(err);
        hll_cleanup(&sm);
        return -1;
    }
    double *d_result;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(&d_result, sm.num_rows * sizeof(double));
#else
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
#endif
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
        return 1;
    }
    double *d_v;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(&d_v, sm.num_rows * sizeof(double));
#else
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
#endif
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    float *samples = (float*) malloc(runs_num * sizeof(float));
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int threads_num = 1024;
    int blocks_num = (int)ceil(sm.num_rows / (double)threads_num);
    for (int i = 0; i < runs_num; ++i) {
        hipEventRecord(start);
        cuda_spmv_hll<<<blocks_num, threads_num>>>(d_result, sm.hack_size, sm.hacks_num, d_data, d_offsets, d_col_index, d_maxnzr, d_v, sm.num_rows);
        hipEventRecord(stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            pr_err(err);
            hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_maxnzr);
            hipFree(d_result);
            hipFree(d_v);
            hll_cleanup(&sm);
            return -1;
        }
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        samples[i] = m;
        sum += m;
    }
    ti->millis = sum / runs_num;
    ti->millis = sum / runs_num;   
    ti->dev = std_dev(samples, ti->millis, runs_num);
    ti->flops = (2 * nz) / ti->millis;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_maxnzr);
    hipFree(d_result);
    hipFree(d_v);
    free(v);
    hll_cleanup(&sm);
    return 0;
}
