#include "cuda_time.h"
#include "hll.h"
#include "spmv_cuda.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define pr_err(err) printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err))

hipError_t cuda_hll_init(struct hll *hll, double **data, int **col_index, int **maxnzr, int **offsets) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * hll->data_num);
    if (err != hipSuccess) {
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * hll->data_num);
    if (err != hipSuccess) {
        hipFree(*data);
        return err;
    }
    err = hipMalloc(maxnzr, sizeof(int) * hll->hacks_num);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMalloc(offsets, sizeof(int) * hll->offsets_num);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    err = hipMemcpy(*data, hll->data, hll->data_num * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*col_index, hll->col_index, hll->data_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*maxnzr, hll->max_nzr, hll->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*offsets, hll->offsets, hll->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    return err;
}

int hll_time(const char *path, int hack_size, struct time_info *ti, int runs_num) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct hll sm;
    if (hll_init(&sm, hack_size, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *v = d_random(mm.num_rows);
    double *d_data;
    int *d_col_index;
    int *d_maxnzr;
    int *d_offsets;
    hipError_t err = cuda_hll_init(&sm, &d_data, &d_col_index, &d_maxnzr, &d_offsets);
    if (err != hipSuccess) {
        pr_err(err);
        hll_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < nruns; ++i) {
        hipEventRecord(start);
        cuda_spmv_hll<<<2, 1024>>>(d_result, sm.hack_size, sm.hacks_num, d_data, d_offsets, d_col_index, d_maxnzr, d_v, m);
        hipEventRecord(stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            pr_err(err);
            hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_maxnzr);
            hipFree(d_result);
            hipFree(d_v);
            hll_cleanup(&sm);
            return -1;
        }
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        sum += m;
    }
    ti->millis = sum / runs_num;
    ti->flops = (2 * nz) / ti->millis;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_maxnzr);
    hipFree(d_result);
    hipFree(d_v)
    free(v);
    hll_cleanup(&sm);
}