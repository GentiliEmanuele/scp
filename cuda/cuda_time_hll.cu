#include "hip/hip_runtime.h"
#include "cuda_time.h"
#include "cuda_mtx.h"
#include "hll.h"
#include "spmv_cuda.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

#define pr_err(err) printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err))

int hll_time(const char *path, int runs_num, int hack_size, struct time_info *ti) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct hll sm;
    if (hll_init(&sm, hack_size, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *v = d_random(mm.num_rows);
    double *d_data;
    int *d_col_index;
    int *d_maxnzr;
    int *d_offsets;
    hipError_t err = cuda_hll_init(&sm, &d_data, &d_col_index, &d_maxnzr, &d_offsets);
    if (err != hipSuccess) {
        pr_err(err);
        hll_cleanup(&sm);
        return -1;
    }
    double *d_result;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(&d_result, sm.num_rows * sizeof(double));
#else
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
#endif
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
        return 1;
    }
    double *d_v;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(&d_v, sm.num_rows * sizeof(double));
#else
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
#endif
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        pr_err(err);
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    float *samples = (float*) malloc(runs_num * sizeof(float));
    if (!samples) {
        printf("out of memory\n");
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    float min = 1e18;
    float max = -1;
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int threads_num = 1024;
    int blocks_num = (int)ceil(sm.hacks_num / (double)threads_num);
    for (int i = 0; i < runs_num; ++i) {
        hipEventRecord(start);
        cuda_spmv_hll_v2<<<blocks_num, threads_num>>>(d_result, sm.hack_size, sm.hacks_num, d_data, d_offsets, d_col_index, d_maxnzr, d_v, sm.num_rows);
        hipEventRecord(stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            pr_err(err);
            hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_maxnzr);
            hipFree(d_result);
            hipFree(d_v);
            hll_cleanup(&sm);
            return -1;
        }
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        samples[i] = m;
        if (m > max) {
            max = m;
        }
        if (m < min) {
            min = m;
        }
        sum += m;
    }
    ti->millis = sum / runs_num;
    ti->millis = sum / runs_num;   
    ti->dev = std_dev(samples, ti->millis, runs_num);
    ti->flops = (2 * nz) / ti->millis;
    ti->min = min;
    ti->max = max;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_maxnzr);
    hipFree(d_result);
    hipFree(d_v);
    free(v);
    hll_cleanup(&sm);
    free(samples);
    return 0;
}
