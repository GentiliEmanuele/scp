#include "cuda_test.h"
#include "spmv_cuda.h"
#include "cuda_mtx.h"
#include "hll.h"
#include "spmv_openmp.h"
#include "utils.h"
#include "vec.h"
#include <hip/hip_runtime.h>

int hll_test(char *path, int hack_size) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct hll sm;
    if (hll_init(&sm, hack_size, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    mtx_cleanup(&mm);
    double *v = d_random(mm.num_rows);
    double *d_data;
    int *d_col_index;
    int *d_maxnzr;
    int *d_offsets;
    hipError_t err = cuda_hll_init(&sm, &d_data, &d_col_index, &d_maxnzr, &d_offsets);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hll_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    cuda_spmv_hll<<<1024, 1024>>>(d_result, sm.hack_size, sm.hacks_num, d_data, d_offsets, d_col_index, d_maxnzr, d_v, sm.num_rows);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
        hipFree(d_col_index);
        hipFree(d_maxnzr);
        hipFree(d_result);
        hipFree(d_v);
        hll_cleanup(&sm);
        return -1;
    }
    double *result = d_zeros(sm.num_rows);
    if (result == NULL) {
        printf("cannot allocate result for cuda\n");
    }
    err = hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    }
    double *test_result = d_zeros(sm.num_rows);
    if (spmv_hll_par(test_result, &sm, v, sm.num_rows)) {
        printf("spmv_hll_par failed\n");
    } else if (!d_veceq(result, test_result, sm.num_rows, 1e-6)) {
        printf("matrix %s\n", path);
        printf("kernel(%d, %d)\n", blocks_num, threads_num);
        printf("test failed\n");
    }
    free(test_result);
    free(result);
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_maxnzr);
    hipFree(d_result);
    hipFree(d_v);
    free(v);
    hll_cleanup(&sm);
    return 0;
}
