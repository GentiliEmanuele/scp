#include "cuda_mtx.h"

hipError_t cuda_csr_init(struct csr *csr, double **data, int **col_index, int **row_pointer) {
    hipError_t err;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(data, sizeof(double) * csr->num_data);
#else
    err = hipMalloc(data, sizeof(double) * csr->num_data);
#endif
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for data\n");
        return err;
    }
#ifdef CUDA_MANAGED
    err = hipMallocManaged(col_index, sizeof(int) * csr->num_data);
#else
    err = hipMalloc(col_index, sizeof(int) * csr->num_data);
#endif
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for col_index\n");
        hipFree(*data);
        return err;
    }
#ifdef CUDA_MANAGED
    err = hipMallocManaged(row_pointer, sizeof(int) * (csr->num_rows + 1));
#else
    err = hipMalloc(row_pointer, sizeof(int) * (csr->num_rows + 1));
#endif
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for row_pointer\n");
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*data, csr -> data, csr->num_data * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*col_index, csr->col_index, csr->num_data * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*row_pointer, csr->row_pointer, (csr->num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    return hipSuccess;
}

hipError_t cuda_hll_init(struct hll *hll, double **data, int **col_index, int **maxnzr, int **offsets) {
    hipError_t err;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(data, sizeof(double) * hll->data_num);
#else
    err = hipMalloc(data, sizeof(double) * hll->data_num);
#endif
    if (err != hipSuccess) {
         printf("cannot allocate enough memory for data\n");
        return err;
    }
#ifdef CUDA_MANAGED
    err = hipMallocManaged(col_index, sizeof(int) * hll->data_num);
#else
    err = hipMalloc(col_index, sizeof(int) * hll->data_num);
#endif
    if (err != hipSuccess) {
         printf("cannot allocate enough memory for col_index\n");
        hipFree(*data);
        return err;
    }
#ifdef CUDA_MANAGED
    err = hipMallocManaged(maxnzr, sizeof(int) * hll->hacks_num);
#else
    err = hipMalloc(maxnzr, sizeof(int) * hll->hacks_num);
#endif
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for maxnzr\n");
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
#ifdef CUDA_MANAGED
    err = hipMallocManaged(offsets, sizeof(int) * hll->offsets_num);
#else
    err = hipMalloc(offsets, sizeof(int) * hll->offsets_num);
#endif
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for offsets\n");
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    err = hipMemcpy(*data, hll->data, hll->data_num * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*col_index, hll->col_index, hll->data_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*maxnzr, hll->max_nzr, hll->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*offsets, hll->offsets, hll->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    return err;
}