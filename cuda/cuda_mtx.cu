#include "cuda_mtx.h"

hipError_t cuda_csr_init(struct csr *csr, double **data, int **col_index, int **row_pointer) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * csr->num_data);
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for data\n");
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * csr->num_data);
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for col_index\n");
        hipFree(*data);
        return err;
    }
    err = hipMalloc(row_pointer, sizeof(int) * (csr->num_rows + 1));
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for row_pointer\n");
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*data, csr -> data, csr->num_data * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*col_index, csr->col_index, csr->num_data * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*row_pointer, csr->row_pointer, (csr->num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    return hipSuccess;
}

hipError_t cuda_hll_init(struct hll *hll, double **data, int **col_index, int **maxnzr, int **offsets) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * hll->data_num);
    if (err != hipSuccess) {
         printf("cannot allocate enough memory for data\n");
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * hll->data_num);
    if (err != hipSuccess) {
         printf("cannot allocate enough memory for col_index\n");
        hipFree(*data);
        return err;
    }
    err = hipMalloc(maxnzr, sizeof(int) * hll->hacks_num);
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for maxnzr\n");
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMalloc(offsets, sizeof(int) * hll->offsets_num);
    if (err != hipSuccess) {
        printf("cannot allocate enough memory for offsets\n");
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    err = hipMemcpy(*data, hll->data, hll->data_num * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*col_index, hll->col_index, hll->data_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*maxnzr, hll->max_nzr, hll->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*offsets, hll->offsets, hll->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    return err;
}