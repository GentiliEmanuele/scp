#include <hip/hip_runtime.h>
#include <stdio.h>

int main(int argc, char *argv) {
    int n = 124406070/3;
    double *d1, *d2, *d3;
    hipError_t err = hipMalloc(&d1, n * sizeof(double));
    if (err != hipSuccess) {
        printf("cannot allocate enough space for d1\n"):
    }
    hipError_t err = hipMalloc(&d2, n * sizeof(double));
    if (err != hipSuccess) {
        printf("cannot allocate enough space for d2\n"):
    }
    hipError_t err = hipMalloc(&d3, n * sizeof(double));
    if (err != hipSuccess) {
        printf("cannot allocate enough space for d3\n"):
    }
    hipFree(*d1);
    hipFree(*d2);
    hipFree(*d3);
}