#include "hip/hip_runtime.h"
#include "spmv_cuda.h"

#define num_of_rows(h, hack_size, hacks_num, num_rows) ((hacks_num - 1 == h && num_rows % hack_size) ? num_rows % hack_size : hack_size)

__global__ void cuda_spmv_hll(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int h = blockDim.x * blockIdx.x + threadIdx.x;
    if (h < hacks_num - 1) {
    	int rows = hack_size;
    	for (int r = 0; r < hack_size; ++r) {
        	double sum = 0.0;
        	for (int j = 0; j < max_nzr[h]; ++j) {
            		int k = offsets[h] + r * max_nzr[h] + j;
            		sum += data[k] * v[col_index[k]];
        	}
        	res[rows * h + r] = sum;
    	}
    } else if (h == hacks_num - 1) {
        int rows = n % hack_size;
        for (int r = 0; r < n % hack_size; ++r) {
        	double sum = 0.0;
        	for (int j = 0; j < max_nzr[h]; ++j) {
            		int k = offsets[h] + r * max_nzr[h] + j;
            		sum += data[k] * v[col_index[k]];
        	}
        	res[rows * h + r] = sum;
    	}
    }
}

__global__ void cuda_spmv_csr(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}

