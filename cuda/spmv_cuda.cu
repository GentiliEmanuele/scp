#include "hip/hip_runtime.h"
#include "spmv_cuda.h"

__global__ void cuda_spmv_hll(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int h = blockDim.x * blockIdx.x + threadIdx.x;
    if (h < hacks_num) {
    	int rows = num_of_rows(0, hack_size, hacks_num, n);
    	for (int r = 0; r < num_of_rows(h, hack_size, hacks_num, n); ++r) {
        	double sum = 0.0;
        	for (int j = 0; j < max_nzr[h]; ++j) {
            		int k = offsets[h] + r * max_nzr[h] + j;
            		sum += data[k] * v[col_index[k]];
        	}
        	res[rows * h + r] = sum;
    	}
    }
}

__global__ void cuda_spmv_csr(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}

