#include "hip/hip_runtime.h"
#include "spmv_cuda.h"
#include <stdio.h>

#define num_of_rows(h, hack_size, hacks_num, num_rows) ((hacks_num - 1 == h && num_rows % hack_size) ? num_rows % hack_size : hack_size)
#define get_hack(row, num_rows, hacks_num) (row) / (num_rows / hacks_num)

__global__ void cuda_spmv_hll(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int h = blockDim.x * blockIdx.x + threadIdx.x;
    if (h < hacks_num) {
    	int rows = num_of_rows(0, hack_size, hacks_num, n);
    	for (int r = 0; r < num_of_rows(h, hack_size, hacks_num, n); ++r) {
        	double sum = 0.0;
        	for (int j = 0; j < max_nzr[h]; ++j) {
            		int k = offsets[h] + r * max_nzr[h] + j;
            		sum += data[k] * v[col_index[k]];
        	}
        	res[rows * h + r] = sum;
    	}
    }
}

__global__ void cuda_spmv_hll_v2(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id / 32;
    int lane = thread_id % 32;
    int row = warp_id;
    double sum = 0.0;
    if (row < n) {
        int hack = get_hack(row, n, hacks_num);
        int row_start = row * max_nzr[hack] + offsets[hack];
        int row_end = (row + 1) * max_nzr[hack] + offsets[hack];
        if (get_hack(row + 1, n, hacks_num) != hack) {
            row_end = offsets[get_hack(row + 1, n, hacks_num)];
        }
        for (int element = row_start + lane; element < row_end; element += 32) {
            sum += data[element] * v[col_index[element]];
        } 
        for (int offset = 16; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }
        if (lane == 0) res[row] = sum;
    }
}

__global__ void cuda_spmv_csr(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}


__global__ void cuda_spmv_csr_v2(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id / 32;
    int lane = thread_id % 32;
    int row = warp_id;
    double sum = 0.0;
    if (row < n) {
        int row_start = row_pointer[row];
        int row_end = row_pointer[row + 1];
        for (int element = row_start + lane; element < row_end; element += 32) {
            sum += data[element] * v[col_index[element]];
        }
        for (int offset = 16; offset > 0; offset /= 2) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }
        if (lane == 0) res[row] = sum;
    }
}

