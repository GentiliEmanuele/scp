#include "hip/hip_runtime.h"
#include "csr.h"
#include "spmv_openmp.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void product(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}

void print_vec(double *v, int n) {
	for (int i = 0; i < n; i++) {
		printf("%d %lg\n", i, v[i]);
	}
}

int cuda_csr_init(struct csr *csr, double **data, int **col_index, int **row_pointer) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * csr -> num_data);
    if (err != hipSuccess) {
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * csr ->num_data);
    if (err != hipSuccess) {
        hipFree(*data);
        return err;
    }
    err = hipMalloc(row_pointer, sizeof(int) * (csr -> num_rows + 1));
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*data, sm.data, csr->num_data * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*col_index, csr->col_index, csr->num_data * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*row_pointer, csr->row_pointer, (csr->num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    return hipSuccess;
}

int main(int argc, char **argv) {
    if (--argc != 1) {
        printf("see usage: program matrix_path\n");
        return -1;
    }
    struct MatrixMarket mm;
    if (read_mtx(argv[1], &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    mtx_cleanup(&mm);
    int m = sm.num_rows;
    double *v = d_random(m);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipError_t err = cuda_csr_init(&sm, &d_data, &d_col_index, &d_row_pointer);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        csr_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	csr_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	csr_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	csr_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    // Perform SAXPY on 1M elements
    // 1 number of block in the grid
    // m number of the thread in the block
    product<<<2, 1024>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, m);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	    hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
    	csr_cleanup(&sm);
        return 1;
    }
    double *result = d_zeros(m);
    hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    char r_path[256];
    sprintf(r_path, "%s.result", argv[1]);
    double *py_result = d_zeros(m);
    if (!spmv_csr_par(py_result, &sm, v, sm.m)) {
        printf("cannot execute csr product\n");
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
    	csr_cleanup(&sm);
        return 1;
    }
    if (!d_veceq(result, py_result, m)) {
        printf("test failed\n");
    } else {
        printf("test passed\n");
    }
    print_vec(result, 10);
    print_vec(py_result, 10);
    free(py_result);
    free(result);
    free(v);
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_row_pointer);
    hipFree(d_result);
    csr_cleanup(&sm);
}
