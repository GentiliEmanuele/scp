#include "hip/hip_runtime.h"
#include "csr.h"
#include "hll.h"
#include "spmv_openmp.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define num_of_rows(h, hack_size, hacks_num, num_rows) ((hacks_num - 1 == h && num_rows % hack_size) ? num_rows % hack_size : hack_size)

__global__ void cuda_spmv_csr(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}

__global__ void cuda_spmv_hll(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int h = blockDim.x * blockIdx.x + threadIdx.x;
    int rows = num_of_rows(0, hack_size, hacks_num, n);
    for (int r = 0; r < num_of_rows(h, hack_size, hacks_num, n); ++r) {
        double sum = 0.0;
        for (int j = 0; j < max_nzr[h]; ++j) {
            int k = offsets[h] + r * max_nzr[h] + j;
            sum += data[k] * v[col_index[k]];
        }
        res[rows * h + r] = sum;
    }
}

void print_vec(double *v, int n) {
	for (int i = 0; i < n; i++) {
		printf("%d %lg\n", i, v[i]);
	}
}

hipError_t cuda_hll_init(struct hll *hll, double **data, int **col_index, int **maxnzr, int **offsets) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * hll->data_num);
    if (err != hipSuccess) {
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * hll->data_num);
    if (err != hipSuccess) {
        hipFree(*data);
        return err;
    }
    err = hipMalloc(maxnzr, sizeof(int) * hll->hacks_num);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMalloc(offsets, sizeof(int) * hll->offsets_num);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    err = hipMemcpy(*data, hll->data, hll->data_num * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*col_index, hll->col_index, hll->data_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*maxnzr, hll->max_nzr, hll->hacks_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*offsets, hll->offsets, hll->offsets_num * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*maxnzr);
    }
    return err;
}

hipError_t cuda_csr_init(struct csr *csr, double **data, int **col_index, int **row_pointer) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * csr -> num_data);
    if (err != hipSuccess) {
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * csr ->num_data);
    if (err != hipSuccess) {
        hipFree(*data);
        return err;
    }
    err = hipMalloc(row_pointer, sizeof(int) * (csr -> num_rows + 1));
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*data, csr -> data, csr->num_data * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*col_index, csr->col_index, csr->num_data * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*row_pointer, csr->row_pointer, (csr->num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    return hipSuccess;
}

int main(int argc, char **argv) {
    if (--argc != 1) {
        printf("see usage: program matrix_path\n");
        return -1;
    }
    struct MatrixMarket mm;
    if (read_mtx(argv[1], &mm)) {
        return -1;
    }
    struct hll sm;
    if (hll_init(&sm, 32, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    mtx_cleanup(&mm);
    int m = sm.num_rows;
    double *v = d_random(m);
    double *d_data;
    int *d_col_index;
    int *d_maxnzr;
    int *d_offsets;
    hipError_t err = cuda_hll_init(&sm, &d_data, &d_col_index, &d_maxnzr, &d_offsets);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hll_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        return 1;        
    }
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hll_cleanup(&sm);
    	hipFree(d_result);
        hipFree(d_v);
        return 1;
    }
    // Perform SAXPY on 1M elements
    // 1 number of block in the grid
    // m number of the thread in the block
    cuda_spmv_hll<<<2, 1024>>>(d_result, sm.hack_size, sm.hacks_num, d_data, d_offsets, d_col_index, d_maxnzr, d_v, m);
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	    hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hipFree(d_result);
        hipFree(d_v);
    	hll_cleanup(&sm);
        return 1;
    }
    double *result = d_zeros(m);
    hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    double *py_result = d_zeros(m);
    if (spmv_hll_par(py_result, &sm, v, m)) {
        printf("cannot execute csr product\n");
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_maxnzr);
    	hipFree(d_result);
    	hll_cleanup(&sm);
        return 1;
    }
    if (!d_veceq(result, py_result, m, 1e-6)) {
        printf("test failed\n");
    } else {
        printf("test passed\n");
    }
    printf("Result \n");
    print_vec(result, 10);
    printf("Pyresult \n");
    print_vec(py_result, 10);
    printf("v\n");
    print_vec(v, 10);
    free(py_result);
    free(result);
    free(v);
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_result);
    hll_cleanup(&sm);
}
