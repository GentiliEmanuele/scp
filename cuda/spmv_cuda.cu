#include "hip/hip_runtime.h"
#include "spmv_cuda.h"
#include <stdio.h>

#define num_of_rows(h, hack_size, hacks_num, num_rows) ((hacks_num - 1 == h && num_rows % hack_size) ? num_rows % hack_size : hack_size)


__global__ void cuda_spmv_hll_v0(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int h = blockDim.x * blockIdx.x + threadIdx.x;
    if (h < hacks_num) {
    	int rows = num_of_rows(0, hack_size, hacks_num, n);
    	for (int r = 0; r < num_of_rows(h, hack_size, hacks_num, n); ++r) {
        	double sum = 0.0;
        	for (int j = 0; j < max_nzr[h]; ++j) {
            		int k = offsets[h] + r * max_nzr[h] + j;
            		sum += data[k] * v[col_index[k]];
        	}
        	res[rows * h + r] = sum;
    	}
    }
}

__global__ void cuda_spmv_hll_v1(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int i = blockDim.x *blockIdx.x + threadIdx.x;
    if (i < n) {
        int hack = i / hack_size;
        int row_start = (i % hack_size) * max_nzr[hack] + offsets[hack];
        int row_end = row_start + max_nzr[hack];
        double sum = 0.0;
        for (int j = row_start; j < row_end; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}

__global__ void cuda_spmv_hll_v2(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ double vTile[];
    if (threadIdx.x < n) vTile[threadIdx.x] = v[threadIdx.x];
    if (i < n) {
        int hack = i / hack_size;
        int row_start = (i % hack_size) * max_nzr[hack] + offsets[hack];
        int row_end = row_start + max_nzr[hack];
        double sum = 0.0;
        for (int j = row_start; j < row_end; ++j) {
            col_index[j] < 1024 ? sum += data[j] * vTile[col_index[j]] : sum += data[j] * v[col_index[j]];
        }
    }
    	res[i] = sum;
}

__global__ void cuda_spmv_hll_v3(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id >> 5;
    int lane = thread_id & 31;
    if (warp_id < n) {
        double sum = 0.0;
        int hack = warp_id / hack_size;
        int row_start = (warp_id % hack_size) * max_nzr[hack] + offsets[hack];
        int row_end = row_start + max_nzr[hack];
        for (int element = row_start + lane; element < row_end; element += 32) {
            sum += data[element] * v[col_index[element]];
        } 
        for (int offset = 16; offset > 0; offset >>= 1) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }
        if (lane == 0) res[warp_id] = sum;
    }
}

__global__ void cuda_spmv_hll_v4(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id >> 5;
    int lane = thread_id & 31;
    extern __shared__ double results[];
    if (warp_id < n) {
        double sum = 0.0;
        int hack = warp_id / hack_size;
        int row_start = (warp_id % hack_size) * max_nzr[hack] + offsets[hack];
        int row_end = row_start + max_nzr[hack];
	    results[threadIdx.x] = 0.0;
        for (int element = row_start + lane; element < row_end; element += 32) {
            results[threadIdx.x] += data[element] * v[col_index[element]];
        } 
	    sum = results[threadIdx.x];
	    for (int offset = 16; offset > 0; offset >>=1) {
            sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
        }
        if (lane == 0) res[warp_id] = sum;
    }
}


__global__ void cuda_spmv_csr(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}


__global__ void cuda_spmv_csr_v2(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    extern __shared__ double vTile[];
    if (threadIdx.x < n) vTile[threadIdx.x] = v[threadIdx.x];
    if (i < n) {
	    double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            col_index[j] < 1024 ? sum += data[j] * vTile[col_index[j]] : sum += data[j] * v[col_index[j]];
        }
	    res[i] = sum;
    }
}


__global__ void cuda_spmv_csr_v3(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id >> 5;
    int lane = thread_id & 31;
    double sum = 0.0;
    if (warp_id > n) return;
    int row_start = row_pointer[warp_id];
    int row_end = row_pointer[warp_id + 1];
    for (int element = row_start + lane; element < row_end; element += 32) {
	    sum += data[element] * v[col_index[element]];
    }
    for (int offset = 16; offset > 0; offset /= 2) {
        sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    }
    if (lane == 0) res[warp_id] = sum;
}


__global__ void cuda_spmv_csr_v4(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id >> 5;
    int lane = thread_id & 31;
    double sum = 0.0;
    extern __shared__ double results[];
    if (warp_id < n) {
	    int row_start = row_pointer[warp_id];
    	int row_end = row_pointer[warp_id + 1];
        results[threadIdx.x] = 0.0;
    	for (int element = row_start + lane; element < row_end; element += 32) {
		    results[threadIdx.x] += data[element] * v[col_index[element]];
    	}
	    sum = results[threadIdx.x];
    	for (int offset = 16; offset > 0; offset /= 2) {
        	sum += __shfl_down_sync(0xFFFFFFFF, sum, offset);
    	}
    	if (lane == 0) res[warp_id] = sum;
    }
}

