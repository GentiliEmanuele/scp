#include "hip/hip_runtime.h"
#include "spmv_cuda.h"

#define num_of_rows(h, hack_size, hacks_num, num_rows) ((hacks_num - 1 == h && num_rows % hack_size) ? num_rows % hack_size : hack_size)

__global__ void cuda_spmv_hll(double *res, int hack_size, int hacks_num, double *data, int *offsets, int *col_index,  int *max_nzr, double *v, int n) {
    int h = blockDim.x * blockIdx.x + threadIdx.x;
    if (h < hacks_num) {
    	int rows = num_of_rows(0, hack_size, hacks_num, n);
    	for (int r = 0; r < num_of_rows(h, hack_size, hacks_num, n); ++r) {
        	double sum = 0.0;
        	for (int j = 0; j < max_nzr[h]; ++j) {
            		int k = offsets[h] + r * max_nzr[h] + j;
            		sum += data[k] * v[col_index[k]];
        	}
        	res[rows * h + r] = sum;
    	}
    }
}

__global__ void cuda_spmv_csr(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = row_pointer[i]; j < row_pointer[i+1]; ++j) {
            sum += data[j] * v[col_index[j]];
        }
        res[i] = sum;
    }
}

__global__ void cuda_spmv_csr_v2(double *res, int *row_pointer, double *data, int *col_index,  double *v, int n) {
    int thread_id = blockDim.x * blockIdx.x + threadIdx.x;
    int warp_id = thread_id / 32;
    int lane = thread_id % 32;
    int row = warp_id;
    double sum = 0.0;
    if (row < n_rows) {
        int row_start = row_pointer[row];
        int row_end = row_pointer[row + 1];
        for (int element = row_start + lane; element < row_end; element += 32) {
            sum += data[element] * x[col_index[element]];
        }
    }
    sum = warp_reduce(sum);
    if (lane == 0 && row < n_rows) {
        res[row] = sum;
    }
}