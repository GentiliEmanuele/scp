#include "hip/hip_runtime.h"
#include "csr.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void product(struct csr *csr, double *res,  double *v, int n) {
    int i = blockIdx.x + threadIdx.x;
    if (i < n) {
        double sum = 0.0;
        for (int j = csr->row_pointer[i]; j < csr->row_pointer[i+1]; ++j) {
            sum += csr->data[j] * v[csr->col_index[j]];
        }
        res[i] = sum;
    }
}


int main(int argc, char **argv) {
    if (--argc != 1) {
        printf("see usage: program matrix_path\n");
        return -1;
    }
    struct MatrixMarket mm;
    if (read_mtx(argv[1], &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return 1;
    }
    int m = sm.num_rows;
    int n = sm.num_cols;
    double *v = d_zeros(m);
    char v_path[256];
    sprintf(v_path, "%s.vector", argv[1]);
    read_vector(v, m, v_path);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipMalloc(&d_data, sizeof(double) * sm.num_data);
    hipMalloc(&d_col_index, sizeof(int) * sm.num_data);
    hipMalloc(&d_row_pointer, sizeof(int) * (sm.num_rows + 1));
    double *d_result;
    hipMalloc(&d_result, sm.num_rows * sizeof(double));
    double *d_v;
    hipMalloc(&d_v, sm.num_rows * sizeof(double));
    hipMemcpy(d_data, sm.data, sm.num_data * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_col_index, sm.col_index, sm.num_data * sizeof(int), hipMemcpyHostToDevice);    hipMemcpy(d_row_pointer, sm.row_pointer, (sm.num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);

    // Perform SAXPY on 1M elements
    // (N+255)/256 number of block in the grid
    // 256 number of the thread in the block
    int N = 1<<20;
    product<<<1, m>>>(&sm, d_result, v, n);
    double *result = d_zeros(m);
    hipMemcpy(d_result, result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    char r_path[256];
    sprintf(r_path, "%s.result", argv[1]);
    double *py_result = d_zeros(m);
    read_vector(py_result, m, r_path);
    if (!d_veceq(py_result, result, m, 1e-6)) {
        printf("test failed!\n");
    } else printf("test passed \n");
    
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_row_pointer);
    hipFree(d_result);
    free(py_result);
    free(result);
    free(v);
    csr_cleanup(&sm);
    mtx_cleanup(&mm);
}
