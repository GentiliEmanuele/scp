#include "csr.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_time.h"
#include "spmv_cuda.h"

hipError_t cuda_csr_init(struct csr *csr, double **data, int **col_index, int **row_pointer) {
    hipError_t err;
    err = hipMalloc(data, sizeof(double) * csr -> num_data);
    if (err != hipSuccess) {
        return err;
    }
    err = hipMalloc(col_index, sizeof(int) * csr ->num_data);
    if (err != hipSuccess) {
        hipFree(*data);
        return err;
    }
    err = hipMalloc(row_pointer, sizeof(int) * (csr -> num_rows + 1));
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        return err;
    }
    err = hipMemcpy(*data, csr -> data, csr->num_data * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*col_index, csr->col_index, csr->num_data * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    err = hipMemcpy(*row_pointer, csr->row_pointer, (csr->num_rows + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        hipFree(*data);
        hipFree(*col_index);
        hipFree(*row_pointer);
        return err;
    }
    return hipSuccess;
}

int csr_time(const char *path, float time, int runs_num, struct time_info *ti) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipError_t err = cuda_csr_init(&sm, &d_data, &d_col_index, &d_row_pointer);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        csr_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	csr_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	hipFree(d_result);
    	csr_cleanup(&sm);
        return 1;        
    }
    int m = sm.num_rows;
    double *v = d_random(m);
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < runs_num; i++) {
        hipEventRecord(start);
        cuda_spmv_csr<<<2, 1024>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, m);
        hipEventRecord(stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	        hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_result);
            hipFree(d_row_pointer);
            hipFree(d_v);
            csr_cleanup(&sm);
            return 1;
        }
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        sum += m;
    }
    ti.millis = sum / runs_num;
    ti.flops = (2 * nz) / ti.millis;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_result);
    hipFree(d_row_pointer);
    hipFree(d_v);
    csr_cleanup(&sm);
    free(v);
}

