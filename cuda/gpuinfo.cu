#include <stdio.h>
#include <hip/hip_runtime.h>

void gpuinfo(void) {
    int dev_count;
    hipGetDeviceCount(&dev_count);
    for (int i = 0; i < dev_count; ++i) {
        hipDeviceProp_t p;
        hipGetDeviceProperties(&p, i);
        printf("#%d name=%s major=%d minor=%d\n", i, p.name, p.major, p.minor);
        printf("global memory=    %d\n", p.totalGlobalMem);
        printf("shared memory=    %d\n", p.sharedMemPerBlock);
        printf("constant memory=  %d\n", p.totalConstMem);
        printf("block registers=  %d\n", p.regsPerBlock);
        printf("warp size=        %d\n", p.warpSize);
        printf("threads per block=%d\n", p.maxThreadsPerBlock);
        printf("max block dim=    %d, %d, %d\n", p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
        printf("max grid dim=     %d, %d, %d\n", p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
    }
}

int main(int argc, char *argv[]) {
    gpuinfo();
}