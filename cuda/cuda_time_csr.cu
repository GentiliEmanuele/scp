#include "csr.h"
#include "utils.h"
#include "vec.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "cuda_time.h"
#include "spmv_cuda.h"
#include "cuda_mtx.h"


int csr_time(const char *path, int runs_num, struct time_info *ti) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipError_t err = cuda_csr_init(&sm, &d_data, &d_col_index, &d_row_pointer);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        csr_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	csr_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	hipFree(d_result);
    	csr_cleanup(&sm);
        return 1;        
    }
    int m = sm.num_rows;
    double *v = d_random(m);
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    for (int i = 0; i < runs_num; i++) {
        hipEventRecord(start);
        cuda_spmv_csr<<<2, 1024>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, m);
        hipEventRecord(stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	        hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_result);
            hipFree(d_row_pointer);
            hipFree(d_v);
            csr_cleanup(&sm);
            return 1;
        }
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        sum += m;
    }
    double *result = d_zeros(sm.num_rows);
    err = hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    }
    print_vec(result, 10);
    ti->millis = sum / runs_num;
    ti->flops = (2 * nz) / ti->millis;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_result);
    hipFree(d_row_pointer);
    hipFree(d_v);
    csr_cleanup(&sm);
    free(v);
    return 0;
}

