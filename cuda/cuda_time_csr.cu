#include "hip/hip_runtime.h"
#include "csr.h"
#include "utils.h"
#include "vec.h"
#include "cuda_time.h"
#include "spmv_cuda.h"
#include "cuda_mtx.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_runtime.h>


int csr_time(const char *path, int runs_num, struct time_info *ti) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    int nz = mm.nz;
    mtx_cleanup(&mm);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipError_t err = cuda_csr_init(&sm, &d_data, &d_col_index, &d_row_pointer);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        csr_cleanup(&sm);
        return -1;
    }
    double *d_result;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(&d_result, sm.num_rows * sizeof(double));
#else
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
#endif
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	csr_cleanup(&sm);
        return 1;
    }
    double *d_v;
#ifdef CUDA_MANAGED
    err = hipMallocManaged(&d_v, sm.num_cols * sizeof(double));
#else
    err = hipMalloc(&d_v, sm.num_cols * sizeof(double));
#endif
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	hipFree(d_result);
    	csr_cleanup(&sm);
        return 1;        
    }
    double *v = d_random(sm.num_cols);
    err = hipMemcpy(d_v, v, sm.num_cols * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    float *samples = (float*) malloc(runs_num * sizeof(float));
    if (!samples) {
        printf("out of memory\n");
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    float min = 1e18;
    float max = -1;
    float sum = 0.0;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    int threads_num = 1024;
    int blocks_num = (int)ceil(sm.num_rows / (double)32);
    for (int i = 0; i < runs_num; i++) {
        hipEventRecord(start);
	    #ifdef csr_v1
    	cuda_spmv_csr<<<blocks_num, threads_num>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, sm.num_rows);
    	#endif
    	#ifdef csr_v2
        int shared_mem_size = threads_num * sizeof(double);
    	cuda_spmv_csr_v2<<<blocks_num, threads_num, shared_mem_size>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, sm.num_rows);
    	#endif
    	#ifdef csr_v3
    	cuda_spmv_csr_v3<<<blocks_num, threads_num>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, sm.num_rows);
    	#endif
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float m = 0.0;
        hipEventElapsedTime(&m, start, stop);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	        hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_result);
            hipFree(d_row_pointer);
            hipFree(d_v);
            csr_cleanup(&sm);
            return 1;
        }
        samples[i] = m;
        if (m > max) {
            max = m;
        }
        if (m < min) {
            min = m;
        }
        sum += m;
    }
    double *result = d_zeros(sm.num_rows);
    err = hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    }
    ti->min = min;
    ti->max = max;
    ti->millis = sum / runs_num;    
    ti->dev = std_dev(samples, ti->millis, runs_num);
    ti->flops = (2 * nz) / ti->millis;
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_result);
    hipFree(d_row_pointer);
    hipFree(d_v);
    csr_cleanup(&sm);
    free(v);
    free(samples);
    return 0;
}

