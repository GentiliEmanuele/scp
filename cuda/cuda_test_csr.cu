#include "hip/hip_runtime.h"
#include "cuda_test.h"
#include "spmv_cuda.h"
#include "cuda_mtx.h"
#include "csr.h"
#include "spmv_openmp.h"
#include "utils.h"
#include "vec.h"
#include <hip/hip_runtime.h>
#include <math.h>

#define CSR2 4

int csr_test(const char *path, int type) {
    if (type == CSR2) {
        #define cuda_opt_csr
    }
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    mtx_cleanup(&mm);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipError_t err = cuda_csr_init(&sm, &d_data, &d_col_index, &d_row_pointer);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        csr_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMallocManaged(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	csr_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMallocManaged(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	hipFree(d_result);
    	csr_cleanup(&sm);
        return 1;        
    }
    double *v = d_random(sm.num_cols);
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    int threads_num = 1024;
    int blocks_num = (int)ceil(sm.num_rows * 32 / (double)threads_num);
    #ifdef cuda_opt_csr
    cuda_spmv_csr_v2<<<blocks_num, threads_num>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, sm.num_rows);
    #else
    cuda_spmv_csr<<<blocks_num, threads_num>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, sm.num_rows);
    #endif
    err = hipGetLastError();
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
        hipFree(d_col_index);
        hipFree(d_result);
        hipFree(d_row_pointer);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    double *result = d_zeros(sm.num_rows);
    err = hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    }
    double *test_result = d_zeros(sm.num_rows);
    if (spmv_csr_par(test_result, &sm, v, sm.num_rows, NULL)) {
        printf("spmv_csr_par failed\n");
    } else if (!d_veceq(result, test_result, sm.num_rows, 1e-6)) {
        printf("matrix %s\n", path);
        // printf("kernel(%d, %d)\n", blocks_num, threads_num);
        printf("test failed\n");
    } else {
	printf("Test passed for %s \n", path);
    }
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_result);
    hipFree(d_row_pointer);
    hipFree(d_v);
    csr_cleanup(&sm);
    free(test_result);
    free(result);
    free(v);
    return 0;
}
