#include "cuda_test.h"
#include "cuda_mtx.h"
#include "csr.h"
#include "spmv_openmp.h"
#include "utils.h"
#include "vec.h"
#include <hip/hip_runtime.h>

int csr_test(const char *path) {
    struct MatrixMarket mm;
    if (read_mtx(path, &mm)) {
        return -1;
    }
    struct csr sm;
    if (csr_init(&sm, &mm)) {
        mtx_cleanup(&mm);
        return -1;
    }
    mtx_cleanup(&mm);
    double *d_data;
    int *d_col_index;
    int *d_row_pointer;
    hipError_t err = cuda_csr_init(&sm, &d_data, &d_col_index, &d_row_pointer);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        csr_cleanup(&sm);
        return -1;
    }
    double *d_result;
    err = hipMalloc(&d_result, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	csr_cleanup(&sm);
        return 1;
    }
    double *d_v;
    err = hipMalloc(&d_v, sm.num_rows * sizeof(double));
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
        hipFree(d_row_pointer);
    	hipFree(d_result);
    	csr_cleanup(&sm);
        return 1;        
    }
    double *v = d_random(sm.num_rows);
    err = hipMemcpy(d_v, v, sm.num_rows * sizeof(double), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
        hipFree(d_data);
    	hipFree(d_col_index);
    	hipFree(d_row_pointer);
    	hipFree(d_result);
        hipFree(d_v);
        csr_cleanup(&sm);
        return 1;
    }
    for (int i = 0; i < 1; i++) {
        cuda_spmv_csr<<<2, 1024>>>(d_result, d_row_pointer, d_data, d_col_index, d_v, m);
        err = hipGetLastError();
        if (err != hipSuccess) {
            printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	        hipFree(d_data);
            hipFree(d_col_index);
            hipFree(d_result);
            hipFree(d_row_pointer);
            hipFree(d_v);
            csr_cleanup(&sm);
            return 1;
        }
    }
    double *result = d_zeros(sm.num_rows);
    err = hipMemcpy(result, d_result, sm.num_rows * sizeof(double), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        printf("error %d (%s): %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
    }
    print_vec(result, 10);
    hipFree(d_data);
    hipFree(d_col_index);
    hipFree(d_result);
    hipFree(d_row_pointer);
    hipFree(d_v);
    csr_cleanup(&sm);
    free(v);
    return 0;
}